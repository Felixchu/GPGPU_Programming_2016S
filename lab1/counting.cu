#include "hip/hip_runtime.h"
#include "counting.h"
#include <cstdio>
#include <cassert>
#include <iostream>
#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/sequence.h>
#include <thrust/copy.h> 
#include <thrust/fill.h> 
#include <thrust/replace.h>

using namespace std;

struct fun_trans{
	__host__ __device__ int operator()( const int &x) const {
		if(x==1)
			return 1;
		else
			return 0;
	}
};

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

__global__ void build(const char *text,int *pos, int *buffer, int text_size, int *offset){
	int idx =threadIdx.x + blockDim.x*blockIdx.x;
	if(idx<text_size){
	int level=0;
	int offset_now=0;
	int offset_next=0;
	for(int i=0;i<2;i++){
		level=0;
	if (text[idx]>=int('a') && text[idx]<=int('z'))
		buffer[idx]=1;
	else
		buffer[idx]=0;
	
	while (((idx%2) ==0)){
		offset_now=offset[level];
		level++;
		offset_next=offset[level];
		buffer[offset_next+idx/2]=(buffer[offset_now+idx]>0 && buffer[offset_now+idx+1]>0)?(buffer[offset_now+idx]+buffer[offset_now+idx+1]):0;
		__syncthreads();
		idx=idx/2;
		if(idx==0)
			break;
	}
	}
	idx = threadIdx.x + blockDim.x*blockIdx.x;
	int head=idx;
	level=0;
	while(1){
		offset_now=offset[level];
		if(buffer[offset_now+head]!=0){
			if(head==0){
				break;
			}
				
			else{
				head=(head-1)/2;
				level++;
			}
		}
		else
			break;
	}

	while(level!=0){
		level=level-1;
		offset_now=offset[level];
		if(buffer[offset_now+head*2+1]==0){
			head=head*2+1;
		}
		else{
			head=head*2;
		}
	}

	pos[idx]=(head==0)?idx-head+1:idx-head;
	}
}



void CountPosition(const char *text, int *pos, int text_size)
{
	int Threadperblock = 256;
	int Blockpergird = ((text_size - Threadperblock + 1) / Threadperblock)+2;
	int *buffer;
	int count=0;
	int size=text_size;
	while(size!=0){
		size=size/2;
		count++;
	}
	int *offset;
	int *offset_cu;
	offset=(int *)malloc(50*sizeof(int));
	memset(offset,0,sizeof(int)*50);
	int i=1;
	size=text_size;
	offset[0]=0;
	while(size!=0){
		offset[i]=offset[i-1]+size;
		size=size/2;
		i++;
	}
	hipMalloc(&offset_cu,sizeof(int)*50);
	hipMemset(offset_cu,0,sizeof(int)*50);
	hipMemcpy(offset_cu,offset,sizeof(int)*50,hipMemcpyHostToDevice);
	hipMalloc(&buffer,sizeof(int)*text_size*2);
	hipMemset(buffer,0,sizeof(int)*text_size*2);
	build << <Blockpergird, Threadperblock >> >(text,pos, buffer ,text_size,offset_cu);
	build << <Blockpergird, Threadperblock >> >(text,pos, buffer ,text_size,offset_cu);
	hipFree(buffer);
	hipFree(offset_cu);
	hipDeviceSynchronize();
}

struct fun_2_trans {
	__host__ __device__ bool operator()(const int x) const {
		if (x==1)
			return 1;
		else
			return 0;
	}
};

int ExtractHead(const int *pos, int *head, int text_size)
{
	int *buffer;
	hipMalloc(&buffer, sizeof(int)*text_size); // this is enough
	hipMemset(buffer,0,sizeof(int)*text_size);
	hipMemset(head,0,sizeof(int)*text_size);
	thrust::device_ptr<const int> pos_d(pos);
	thrust::device_ptr<int> head_d(head), flag_d(buffer);
	thrust::sequence(flag_d,flag_d+text_size );
	//thrust::transform(pos_d,pos_d+text_size,flag_d,fun_trans());
	hipDeviceSynchronize();
	thrust::copy_if(flag_d,flag_d+text_size,pos_d,head_d,fun_2_trans());
	hipDeviceSynchronize();
	int * head_pc=(int *)malloc(text_size*sizeof(int));
	memset(head_pc,0,sizeof(int)*text_size);
	hipMemcpy(head_pc,head,sizeof(int)*text_size,hipMemcpyDeviceToHost);
	int i=0;
	while(1){
		i++;
		if(head_pc[i]==0){
			break;
		}
	}
	hipFree(buffer);
	return i;
}

void Part3(char *text, int *pos, int *head, int text_size, int n_head)
{
}

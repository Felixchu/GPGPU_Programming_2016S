#include "hip/hip_runtime.h"
#include "counting.h"
#include <cstdio>
#include <cassert>
#include <iostream>
#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
using namespace std;

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

__global__ void build(const char *text,int *pos, int *buffer, int text_size, int *offset){
	int idx =threadIdx.x + blockDim.x*blockIdx.x;
	if (text[idx]>=int('a') && text[idx]<=int('z'))
		buffer[idx]=1;
	else
		buffer[idx]=0;
	int level=0;
	int offset_now=0;
	int offset_next=0;
	while (((idx%2) ==0)){
		offset_now=offset[level];
		level++;
		offset_next=offset[level];
		buffer[offset_next+idx/2]=(buffer[offset_now+idx]>0 && buffer[offset_now+idx+1]>0)?(buffer[offset_now+idx]+buffer[offset_now+idx+1]):0;
		__syncthreads();
		idx=idx/2;
		if(idx==0)
			break;
	}
	idx = threadIdx.x + blockDim.x*blockIdx.x;
	int head=idx;
	level=0;
	while(1){
		offset_now=offset[level];
		if(buffer[offset_now+head]!=0){
			if(head==0){
				break;
			}
				
			else{
				head=(head-1)/2;
				level++;
			}
		}
		else
			break;
	}

	while(level!=0){
		level=level-1;
		offset_now=offset[level];
		if(buffer[offset_now+head*2+1]==0){
			head=head*2+1;
		}
		else{
			head=head*2;
		}
	}

	pos[idx]=(head==0)?idx-head+1:idx-head;
}



void CountPosition(const char *text, int *pos, int text_size)
{
	int Threadperblock = 256;
	int Blockpergird = (text_size - Threadperblock + 1) / Threadperblock;
	int *buffer;
	int count=0;
	int size=text_size;
	while(size!=0){
		size=size/2;
		count++;
	}
	int *offset;
	int *offset_cu;
	offset=(int *)malloc(50*sizeof(int));
	int i=1;
	size=text_size;
	offset[0]=0;
	while(size!=0){
		offset[i]=offset[i-1]+size;
		size=size/2;
		i++;
	}
	hipMalloc(&offset_cu,sizeof(int)*count);
	hipMemcpy(offset_cu,offset,sizeof(int)*count,hipMemcpyHostToDevice);
	hipMalloc(&buffer,sizeof(int)*text_size*2);
	build << <Blockpergird, Threadperblock >> >(text,pos, buffer ,text_size,offset_cu);
	hipDeviceSynchronize();
	hipFree(buffer);
	hipFree(offset_cu);
}

int ExtractHead(const int *pos, int *head, int text_size)
{
	int *buffer;
	int nhead;
	hipMalloc(&buffer, sizeof(int)*text_size*2); // this is enough
	thrust::device_ptr<const int> pos_d(pos);
	thrust::device_ptr<int> head_d(head), flag_d(buffer), cumsum_d(buffer+text_size);

	// TODO

	hipFree(buffer);
	return nhead;
}

void Part3(char *text, int *pos, int *head, int text_size, int n_head)
{
}

#include "hip/hip_runtime.h"
#include "lab2.h"
#include <stdlib.h>
#include <iostream>
using namespace std;
#define d_vel_x(a,b) d_vel_x[(b)+(a)*blockDim.x*gridDim.x]
#define d_vel_y(a,b) d_vel_y[(b)+(a)*blockDim.x*gridDim.x]
#define d_vel_o_x(a,b) d_vel_o_x[(b)+(a)*blockDim.x*gridDim.x]
#define d_vel_o_y(a,b) d_vel_o_y[(b)+(a)*blockDim.x*gridDim.x]
#define d_vel_t_x(a,b) d_vel_t_x[(b)+(a)*blockDim.x*gridDim.x]
#define d_vel_t_y(a,b) d_vel_t_y[(b)+(a)*blockDim.x*gridDim.x]
#define d_den(a,b) d_den[(b)+(a)*blockDim.x*gridDim.x]
#define d_den_o(a,b) d_den_o[(b)+(a)*blockDim.x*gridDim.x]
#define gpudeclare() int x=threadIdx.x+blockIdx.x*blockDim.x;int y=threadIdx.y+blockIdx.y*blockDim.y;int id=x+y*blockDim.x*gridDim.x;
static const unsigned W = 640;
static const unsigned H = 480;
static const unsigned NFRAME =500;


static const dim3 dimGrid(32,24);
static const dim3 dimBlock(20,20);
struct Lab2VideoGenerator::Impl {
	int t = 0;
};
__global__ void initial(float* d_vel_x,float* d_vel_y,float* d_vel_o_x,float* d_vel_o_y,float* d_vel_t_x,float* d_vel_t_y,float* d_den,float* d_den_o,float* d_den_t){
	int x=threadIdx.x+blockIdx.x*blockDim.x;
	int y=threadIdx.y+blockIdx.y*blockDim.y;
	int id=x+y*blockDim.x*gridDim.x;
	d_den[id]=x*800;
	d_vel_x[id]=0;
	d_vel_y[id]=0;
	d_den_o[id]=800;
	//d_den_o[id]=x*400;
	d_vel_o_x[id]=0;
	d_vel_o_y[id]=0;
	d_vel_t_x[id]=0;
	d_vel_t_y[id]=0;
	d_den_t[id]=0;
	//d_vel_o_x(1,x)=4;
	//d_vel_o_x(100,x)=2;
    //
	//	
	//d_vel_o_x(200,x)=-2;
	//
	if((x-100)*(x-100)+(y-100)*(y-100) <5000){
		d_vel_x[id]=y-100;
		d_vel_y[id]=x-100;
	}
	
	if((x-100)*(x-100)+(y-300)*(y-300) <5000){
		d_vel_o_x[id]=y-300;
		d_vel_o_y[id]=x-100;
	}
	
	if((x-540)*(x-540)+(y-100)*(y-100) <5000){
		d_vel_o_x[id]=y-100;
		d_vel_o_y[id]=x-540;
	}
	if((x-540)*(x-540)+(y-300)*(y-300) <5000){
		d_vel_o_x[id]=y-300;
		d_vel_o_y[id]=x-540;
	}
	
	if((x-200)*(x-200)+(y-100)*(y-100) <5000){
		d_vel_o_x[id]=y-100;
		d_vel_o_y[id]=x-540;
	}
	if((x-300)*(x-300)+(y-300)*(y-300) <5000){
		d_vel_o_x[id]=y-300;
		d_vel_o_y[id]=x-540;
	}
	if((x-400)*(x-400)+(y-100)*(y-100) <5000){
		d_vel_o_x[id]=y-100;
		d_vel_o_y[id]=x-540;
	}
	if((x-500)*(x-500)+(y-300)*(y-300) <5000){
		d_vel_o_x[id]=y-300;
		d_vel_o_y[id]=x-540;
	}
	
}
__global__ void read(uint8_t* yuv,float* d_den,int max_den){
	gpudeclare();
	yuv[id]=(d_den[id]<max_den)?255*d_den[id]/max_den:255;
	//yuv[id]=d_den[id];
}

__global__ void add(float* d_vel_x,float* d_vel_y,float* d_vel_o_x,float* d_vel_o_y,int dt){
	int x=threadIdx.x+blockIdx.x*blockDim.x;
	int y=threadIdx.y+blockIdx.y*blockDim.y;
	int id=x+y*blockDim.x*gridDim.x;
	d_vel_x[id]+=d_vel_o_x[id]*dt;
	d_vel_y[id]+=d_vel_o_y[id]*dt;
}

__global__ void add_den(float* d_den,float* d_den_o,float dt){
	gpudeclare();
	d_den[id]+=d_den_o[id]*dt;
}

__global__ void diffusion(float* d_vel_x,float* d_vel_y,float* d_vel_o_x,float* d_vel_o_y,float*d_vel_t_x ,float* d_vel_t_y,float dt,float viscosity){
	gpudeclare();
	float a= dt*viscosity*W*H;
		if((x>0) && (x<W-1) && (y>=1) && (y<H-1)){
			d_vel_x[id]=(d_vel_o_x[id]+a*(d_vel_o_x(y-1,x)+d_vel_o_x(y+1,x)+d_vel_o_x(y,x-1)+d_vel_o_x(y,x+1))-4*d_vel_o_x(y,x));
			d_vel_y[id]=(d_vel_o_y[id]+a*(d_vel_o_y(y-1,x)+d_vel_o_y(y+1,x)+d_vel_o_y(y,x-1)+d_vel_o_y(y,x+1))-4*d_vel_o_y(y,x));
		}
}

__global__ void diffusion_den(float* d_den,float* d_den_o,float* d_den_t,float dt,float viscosity){
	gpudeclare();
	float a= dt*viscosity*W*H;
		if((x>0) && (x<W-1) && (y>=1) && (y<H-1)){
			d_den[id]=(d_den_o[id]+a*(d_den_o(y-1,x)+d_den_o(y+1,x)+d_den_o(y,x-1)+d_den_o(y,x+1)-4*d_den_o(y,x)));
		}
}

__global__ void set_vel_bound(float* d_vel_x,float* d_vel_y,float* d_vel_t_x,float* d_vel_t_y){
	gpudeclare();
	d_vel_t_x[id]=d_vel_x[id];
	d_vel_t_y[id]=d_vel_y[id];
	if(x==0)
		d_vel_t_y[id]= -d_vel_y(y,1);
	if(x==W-1)
		d_vel_t_y[id]= -d_vel_y(y,W-2);
	if(y==0)
		d_vel_t_x[id]= -d_vel_x(1,x);
	if(y==H-1)
		d_vel_t_x[id]= -d_vel_x(H-2,x);
	if(x==0 && y==0){
		d_vel_t_x[id]=0.5 *(d_vel_x(1,0)+d_vel_x(0,1));
		d_vel_t_y[id]=0.5 *(d_vel_y(1,0)+d_vel_y(0,1));
	}
	if(x==W-1 && y==0){
		d_vel_t_x[id]=0.5 *(d_vel_x(1,W-1)+d_vel_x(0,W-1));
		d_vel_t_y[id]=0.5 *(d_vel_y(1,W-1)+d_vel_y(0,W-1));
	}
	if(x==0 && y==H-1){
		d_vel_t_x[id]=0.5 *(d_vel_x(H-2,0)+d_vel_x(H-1,1));
		d_vel_t_y[id]=0.5 *(d_vel_y(H-2,0)+d_vel_y(H-1,1));
	}
	if(x==W-1 && y==H-1){
		d_vel_t_x[id]= 0.5*(d_vel_x(H-2,W-1)+d_vel_x(H-1,W-2));
		d_vel_t_y[id]= 0.5*(d_vel_y(H-2,W-1)+d_vel_y(H-1,W-2));
	}

	if(y>200 && y<300 && x>200&& x<300){
		d_vel_t_x[id]= -d_vel_x[id];
		d_vel_t_y[id]= -d_vel_y[id];
	}
}
__global__ void set_den_bound(float* d_den,float* d_den_t){
	gpudeclare();
	d_den_t[id]=d_den[id];
	if(x==0)
		d_den_t[id]= -d_den(y,1);
	if(x==W-1)
		d_den_t[id]= -d_den(y,W-2);
	if(y==0)
		d_den_t[id]= -d_den(1,x);
	if(y==H-1)
		d_den_t[id]= -d_den(H-2,x);
	if(x==0 && y==0){
		d_den_t[id]=0.5 *(d_den(1,0)+d_den(0,1));
	}
	if(x==W-1 && y==0){
		d_den_t[id]=0.5 *(d_den(1,W-1)+d_den(0,W-1));
	}
	if(x==0 && y==H-1){
		d_den_t[id]=0.5 *(d_den(H-2,0)+d_den(H-1,1));
	}
	if(x==W-1 && y==H-1){
		d_den_t[id]= 0.5*(d_den(H-2,W-1)+d_den(H-1,W-2));
	}
}


__global__ void GaussSeidel(float* d_vel_x,float* d_vel_y,float*d_vel_o_x ,float* d_vel_o_y,float*d_vel_t_x ,float* d_vel_t_y){
	gpudeclare();
	if((x>0) && (x<W-1) && (y>=1) && (y<H-1)){
		d_vel_o_y[id]= (d_vel_x(y,x) + d_vel_y(y-1,x)+d_vel_y(y+1,x)+d_vel_y(y,x-1)+d_vel_y(y,x+1)-4*d_vel_y(y,x)); 
	}
}

__global__ void project_set_vel(float* d_vel_x,float* d_vel_y,float*d_vel_o_x ,float* d_vel_o_y){
	gpudeclare();
	float h_x=1.0/W;
	float h_y=1.0/H;
	if((x>0) && (x<W-1) && (y>=1) && (y<H-1)){
		d_vel_x[id]=d_vel_x[id]-0.5*(d_vel_o_y(y+1,x)-d_vel_o_y(y-1,x))/h_x;
		d_vel_y[id]=d_vel_y[id]-0.5*(d_vel_o_y(y+1,x)-d_vel_o_y(y-1,x))/h_y;
	}
}

__global__ void project_set_div(float* d_vel_x,float* d_vel_y,float*d_vel_o_x ,float* d_vel_o_y){
	gpudeclare();
	float h_x=1.0/W;
	if((x>0) && (x<W-1) && (y>=1) && (y<H-1)){
		d_vel_o_x[id]=-0.5 * h_x *(d_vel_x(y+1,x)-d_vel_x(y-1,x)+d_vel_y(y,x+1)-d_vel_y(y,x-1));
		d_vel_o_y[id]=0;
	}
}

__global__ void addvection(float* d_vel_x,float*d_vel_y,float*d_vel_o_x,float*d_vel_o_y,float*d_vel_in_x,float*d_vel_in_y,float dt){
	gpudeclare();
	if((x>0) && (x<W-1) && (y>=1) && (y<H-1)){
		float x_i= x-dt*W*d_vel_in_x[id];
		float y_i= y-dt*H*d_vel_in_y[id];
		if(x_i<0.5) x_i=0.5;
		if(x_i>W+0.5) x_i=W+0.5;
		int x_i0=(int) x_i;
		int x_i1=x_i0+1;
		if(y_i<0.5) y_i=0.5;
		if(y_i>W+0.5) y_i=W+0.5;
		int y_i0=(int) y_i;
		int y_i1=y_i0+1;
		float x_i_f = x - x_i0;
		float x_i_fc = 1 - x_i_f;
		float y_i_f = y - y_i0;
		float y_i_fc = 1 - y_i_f;
		
		d_vel_x[id]=x_i_fc *(y_i_fc*d_vel_o_x(y_i0,x_i0)+y_i_f*d_vel_o_x(y_i0,x_i1))+x_i_f*(y_i_fc*d_vel_o_x(y_i1,x_i0)+y_i_f*d_vel_o_x(y_i1,x_i1));
		d_vel_y[id]=x_i_fc *(y_i_fc*d_vel_o_y(y_i0,x_i0)+y_i_f*d_vel_o_y(y_i0,x_i1))+x_i_f*(y_i_fc*d_vel_o_y(y_i1,x_i0)+y_i_f*d_vel_o_y(y_i1,x_i1));
	}
}


__global__ void addvection_den(float* d_den,float*d_den_o,float*d_den_t,float*d_vel_x,float*d_vel_y,float dt){
	gpudeclare();
	if((x>0) && (x<W-1) && (y>=1) && (y<H-1)){
		float x_i= x-dt*W*d_vel_x[id];
		float y_i= y-dt*H*d_vel_y[id];
		if(x_i<0.5) x_i=0.5;
		if(x_i>W+0.5) x_i=W+0.5;
		int x_i0=(int) x_i;
		int x_i1=x_i0+1;
		if(y_i<0.5) y_i=0.5;
		if(y_i>W+0.5) y_i=W+0.5;
		int y_i0=(int) y_i;
		int y_i1=y_i0+1;
		float x_i_f = x - x_i0;
		float x_i_fc = 1 - x_i_f;
		float y_i_f = y - y_i0;
		float y_i_fc = 1 - y_i_f;
		
		d_den[id]=x_i_fc *(y_i_fc*d_den_o(y_i0,x_i0)+y_i_f*d_den_o(y_i0,x_i1))+x_i_f*(y_i_fc*d_den_o(y_i1,x_i0)+y_i_f*d_den_o(y_i1,x_i1));
	}
}

__global__ void action_1(float* d_vel_x,float* d_vel_y,float* d_den,float* d_vel_o_x,float* d_vel_o_y,float* d_den_o){
	gpudeclare();

	d_vel_x[id]=0;
	d_vel_y[id]=0;
	d_vel_o_x[id]=0;
	d_vel_o_y[id]=0;
	if((x-200)*(x-200)+(y-100)*(y-100) <2000){
		d_vel_o_x[id]=y-100;
		d_vel_o_y[id]=x-100;
	}
	
	if((x-200)*(x-200)+(y-300)*(y-300) <2000){
		d_vel_o_x[id]=y-300;
		d_vel_o_y[id]=x-100;
	}
	
	if((x-540)*(x-540)+(y-100)*(y-100) <2000){
		d_vel_o_x[id]=y-100;
		d_vel_o_y[id]=x-540;
	}
	if((x-540)*(x-540)+(y-300)*(y-300) <2000){
		d_vel_o_x[id]=y-300;
		d_vel_o_y[id]=x-540;
	}

}

__global__ void action_2(float* d_vel_x,float* d_vel_y,float* d_den,float* d_vel_o_x,float* d_vel_o_y,float* d_den_o){
	gpudeclare();

	d_vel_x[id]=0;
	d_vel_y[id]=0;
	d_vel_o_x[id]=0;
	d_vel_o_y[id]=0;
	if((x-200)*(x-200)+(y-100)*(y-100) <5000){
		d_vel_o_x[id]=y-100;
		d_vel_o_y[id]=x-100;
	}
	
	if((x-200)*(x-200)+(y-300)*(y-300) <5000){
		d_vel_o_x[id]=y-300;
		d_vel_o_y[id]=x-100;
	}
	
	if((x-540)*(x-540)+(y-100)*(y-100) <5000){
		d_vel_o_x[id]=y-100;
		d_vel_o_y[id]=x-540;
	}
	if((x-540)*(x-540)+(y-300)*(y-300) <5000){
		d_vel_o_x[id]=y-300;
		d_vel_o_y[id]=x-540;
	}

}

__global__ void action_3(float* d_vel_x,float* d_vel_y,float* d_den,float* d_vel_o_x,float* d_vel_o_y,float* d_den_o){
	gpudeclare();
	d_vel_x[id]=0;
	d_vel_y[id]=0;
	d_vel_o_x[id]=0;
	d_vel_o_y[id]=0;
	if((x-200)*(x-200)+(y-100)*(y-100) <200){
		d_vel_o_x[id]=y-100;
		d_vel_o_y[id]=x-100;
	}
	
	if((x-200)*(x-200)+(y-300)*(y-300) <200){
		d_vel_o_x[id]=y-300;
		d_vel_o_y[id]=x-100;
	}
	
	if((x-540)*(x-540)+(y-100)*(y-100) <200){
		d_vel_o_x[id]=y-100;
		d_vel_o_y[id]=x-540;
	}
	if((x-540)*(x-540)+(y-300)*(y-300) <200){
		d_vel_o_x[id]=y-300;
		d_vel_o_y[id]=x-540;
	}
	if((x-400)*(x-400)+(y-100)*(y-100) <200){
		d_vel_o_x[id]=y-100;
		d_vel_o_y[id]=x-400;
	}
	
	if((x-400)*(x-400)+(y-300)*(y-300) <200){
		d_vel_o_x[id]=y-300;
		d_vel_o_y[id]=x-400;
	}
	
	if((x-540)*(x-540)+(y-200)*(y-200) <200){
		d_vel_o_x[id]=y-200;
		d_vel_o_y[id]=x-540;
	}
	if((x-400)*(x-400)+(y-200)*(y-200) <200){
		d_vel_o_x[id]=y-300;
		d_vel_o_y[id]=x-400;
	}
}
__global__ void action_4(float* d_vel_x,float* d_vel_y,float* d_den,float* d_vel_o_x,float* d_vel_o_y,float* d_den_o){
	gpudeclare();

	d_vel_x[id]=0;
	d_vel_y[id]=0;
	d_vel_o_x[id]=0;
	d_vel_o_y[id]=0;
	if((x-300)*(x-300)+(y-200)*(y-200) <1000){
		d_vel_o_x[id]=y-200;
		d_vel_o_y[id]=x-300;
	}
	
	if((x-300)*(x-300)+(y-200)*(y-200) <5000){
		d_vel_o_x[id]=y-200;
		d_vel_o_y[id]=x-300;
	}
	
	if((x-300)*(x-300)+(y-200)*(y-200) <10000){
		d_vel_o_x[id]=y-200;
		d_vel_o_y[id]=x-300;
	}
	if((x-300)*(x-300)+(y-200)*(y-200) <20000){
		d_vel_o_x[id]=y-200;
		d_vel_o_y[id]=x-300;
	}
	
}


__global__ void action_5(float* d_vel_x,float* d_vel_y,float* d_den,float* d_vel_o_x,float* d_vel_o_y,float* d_den_o){
	gpudeclare();

	d_vel_x[id]=0;
	d_vel_y[id]=0;
	d_vel_o_x[id]=0;
	d_vel_o_y[id]=0;
	if((x-200)*(x-200)+(y-100)*(y-100) <2000){
		d_vel_o_x[id]=y-100;
		d_vel_o_y[id]=x-100;
	}
	
	if((x-200)*(x-200)+(y-300)*(y-300) <2000){
		d_vel_o_x[id]=y-300;
		d_vel_o_y[id]=x-100;
	}
	
	if((x-540)*(x-540)+(y-100)*(y-100) <2000){
		d_vel_o_x[id]=y-100;
		d_vel_o_y[id]=x-540;
	}
	if((x-540)*(x-540)+(y-300)*(y-300) <2000){
		d_vel_o_x[id]=y-300;
		d_vel_o_y[id]=x-540;
	}
}


__global__ void action_6(float* d_vel_x,float* d_vel_y,float* d_den,float* d_vel_o_x,float* d_vel_o_y,float* d_den_o){
	gpudeclare();

	d_vel_x[id]=0;
	d_vel_y[id]=0;
	d_vel_o_x[id]=0;
	d_vel_o_y[id]=0;
	if((x<300) && (x>100) && (y<300) && (y>100)){
		d_vel_o_x[id]=100;
		d_vel_o_y[id]=100;
	}
	if((x<400) && (x>300) && (y<300) && (y>100)){
		d_vel_o_x[id]=200;
		d_vel_o_y[id]=200;
	}
	if((x<500) && (x>450) && (y<300) && (y>100)){
		d_vel_o_x[id]=300;
		d_vel_o_y[id]=300;
	}
	if((x<600) && (x>550) && (y<300) && (y>100)){
		d_vel_o_x[id]=400;
		d_vel_o_y[id]=400;
	}
	if((x<300) && (x>100) && (y<400) && (y>350)){
		d_vel_o_x[id]=500;
		d_vel_o_y[id]=500;
	}
	if((x<300) && (x>100) && (y<100) && (y>20)){
		d_vel_o_x[id]=600;
		d_vel_o_y[id]=600;
	}
	
	

}

__global__ void action_7(float* d_vel_x,float* d_vel_y,float* d_den,float* d_vel_o_x,float* d_vel_o_y,float* d_den_o){
	gpudeclare();

	d_vel_x[id]=0;
	d_vel_y[id]=0;
	d_vel_o_x[id]=0;
	d_vel_o_y[id]=0;
	if((x<300) && (x>100) && (y<300) && (y>100)){
		d_vel_o_x[id]=100;
		d_vel_o_y[id]=100;
	}
}



void  Lab2VideoGenerator::swap(float* d_vel_x,float* d_vel_y,float* d_vel_o_x,float* d_vel_o_y){
	float * temp;
	temp=d_vel_x;
	d_vel_x=d_vel_o_x;
	d_vel_o_x=temp;
	temp=d_vel_y;
	d_vel_y=d_vel_o_y;
	d_vel_o_y=temp;
}


void  Lab2VideoGenerator::set_vel_bound_h(float* d_vel_x, float* d_vel_y, float* d_vel_t_x, float* d_vel_t_y){
	hipMemset(d_vel_t_x, 0, W*H);
	hipMemset(d_vel_t_y, 0, W*H);
	set_vel_bound<<<dimGrid,dimBlock>>>(d_vel_x,d_vel_y,d_vel_t_x,d_vel_t_y);
	hipDeviceSynchronize();
	swap(d_vel_x,d_vel_y,d_vel_t_x,d_vel_t_y);
}

void  Lab2VideoGenerator::set_den_bound_h(float* d_den, float* d_den_t){
	hipMemset(d_den_t, 0, W*H);
	set_den_bound<<<dimGrid,dimBlock>>>(d_den,d_den_t);
	hipDeviceSynchronize();
	swap(d_den,NULL,d_den_t,NULL);
}

void  Lab2VideoGenerator::GaussSeidel_h(float* d_vel_x,float* d_vel_y,float* d_vel_o_x,float* d_vel_o_y,float* d_vel_t_x,float* d_vel_t_y){
		GaussSeidel<<<dimGrid,dimBlock>>>(d_vel_x,d_vel_y,d_vel_o_x,d_vel_o_y,d_vel_t_x,d_vel_t_y);
		hipDeviceSynchronize();
		swap(NULL,d_vel_t_y,NULL,d_vel_o_y);
		set_vel_bound_h(d_vel_o_x,d_vel_o_y,d_vel_t_x,d_vel_t_y);
		set_vel_bound_h(d_vel_o_x,d_vel_o_y,d_vel_t_x,d_vel_t_y);
}

void Lab2VideoGenerator::project(){
	project_set_div<<<dimGrid,dimBlock>>>(d_vel_x,d_vel_y,d_vel_o_x,d_vel_o_y);
	hipDeviceSynchronize();
	set_vel_bound_h(d_vel_o_x,d_vel_o_y,d_vel_t_x,d_vel_t_y);
	GaussSeidel_h(d_vel_x,d_vel_y,d_vel_o_x,d_vel_o_y,d_vel_t_x,d_vel_t_y);
	project_set_vel<<<dimGrid,dimBlock>>>(d_vel_x,d_vel_y,d_vel_o_x,d_vel_o_y);
	hipDeviceSynchronize();
	set_vel_bound_h(d_vel_x,d_vel_y,d_vel_t_x,d_vel_t_y);
}

Lab2VideoGenerator::Lab2VideoGenerator(): impl(new Impl) {
	hipMalloc((void**)&d_vel_x,H*W*sizeof(float));
	hipMalloc((void**)&d_vel_y,H*W*sizeof(float));
	hipMalloc((void**)&d_den,H*W*sizeof(float));
	hipMalloc((void**)&d_vel_o_x,H*W*sizeof(float));
	hipMalloc((void**)&d_vel_o_y,H*W*sizeof(float));
	hipMalloc((void**)&d_den_o,H*W*sizeof(float));
	hipMalloc((void**)&d_vel_t_x,H*W*sizeof(float));
	hipMalloc((void**)&d_vel_t_y,H*W*sizeof(float));
	hipMalloc((void**)&d_vel_tt_x,H*W*sizeof(float));
	hipMalloc((void**)&d_vel_tt_y,H*W*sizeof(float));
	hipMalloc((void**)&d_den_t,H*W*sizeof(float));
	initial<<<dimGrid,dimBlock>>>(d_vel_x,d_vel_y,d_vel_o_x,d_vel_o_y,d_vel_t_x,d_vel_t_y,d_den,d_den_o,d_den_t);
	hipDeviceSynchronize();
	dt=0.01;
	viscosity=0.02;
	counter=0;
	max_den=1000;
}

Lab2VideoGenerator::~Lab2VideoGenerator() {}

void Lab2VideoGenerator::get_info(Lab2VideoInfo &info) {
	info.w = W;
	info.h = H;
	info.n_frame = NFRAME;
	// fps = 24/1 = 24
	info.fps_n = 24;
	info.fps_d = 1;
};




void Lab2VideoGenerator::diffusion_h(float* d_vel_x, float* d_vel_y, float* d_vel_o_x, float* d_vel_o_y, float* d_vel_t_x, float* d_vel_t_y){
		diffusion<<<dimGrid,dimBlock>>>(d_vel_x,d_vel_y,d_vel_o_x,d_vel_o_y,d_vel_t_x,d_vel_t_y,dt,viscosity);
		hipDeviceSynchronize();
	set_vel_bound_h(d_vel_x,d_vel_y,d_vel_t_x,d_vel_t_y);
}

void Lab2VideoGenerator::diffusion_den_h(float* d_den, float* d_den_o, float* d_den_t){
		diffusion_den<<<dimGrid,dimBlock>>>(d_den,d_den_o,d_den_t,dt,viscosity);
		hipDeviceSynchronize();
	set_den_bound_h(d_den,d_den_t);
}

void Lab2VideoGenerator::addvection_h(float* d_vel_x, float* d_vel_y, float* d_vel_o_x, float* d_vel_o_y,float* d_vel_in_x,float* d_vel_in_y,float* d_vel_t_x, float* d_vel_t_y){
	addvection<<<dimGrid,dimBlock>>>(d_vel_x,d_vel_y,d_vel_o_x,d_vel_o_y,d_vel_in_x,d_vel_in_y,dt);
	hipDeviceSynchronize();
	set_vel_bound_h(d_vel_x,d_vel_y,d_vel_t_x,d_vel_t_y);
}
void Lab2VideoGenerator::addvection_den_h(float* d_den, float* d_den_o, float* d_den_t, float* d_vel_x,float* d_vel_y){
	addvection_den<<<dimGrid,dimBlock>>>(d_den,d_den_o,d_den_t,d_vel_x,d_vel_y,dt);
	hipDeviceSynchronize();
	set_den_bound_h(d_den,d_den_t);
}

void Lab2VideoGenerator::velocity(){
	add<<<dimGrid,dimBlock>>>(d_vel_x,d_vel_y,d_vel_o_x,d_vel_o_y,dt);
	hipDeviceSynchronize();
	swap(d_vel_x,d_vel_y,d_vel_o_x,d_vel_o_y);
	diffusion_h(d_vel_x,d_vel_y,d_vel_o_x,d_vel_o_y,d_vel_t_x,d_vel_t_y);
	project();
	swap(d_vel_x,d_vel_y,d_vel_o_x,d_vel_o_y);
	addvection_h(d_vel_x,d_vel_y,d_vel_o_x,d_vel_o_y,d_vel_o_x,d_vel_o_y,d_vel_t_x,d_vel_t_y);
	project();
}

void Lab2VideoGenerator::density(){
	add_den<<<dimGrid,dimBlock>>>(d_den,d_den_o,dt);
	swap(d_den,NULL,d_den_o,NULL);
	diffusion_den_h(d_den,d_den_o,d_den_t);
	swap(d_den,NULL,d_den_o,NULL);
	addvection_den_h(d_den,d_den_o,d_den_t,d_vel_x,d_vel_y);
}

void Lab2VideoGenerator::action_h(){
	if(counter==48)
		action_1<<<dimGrid,dimBlock>>>(d_vel_x,d_vel_y,d_den,d_vel_o_x,d_vel_o_y,d_den_o);
	if(counter==96)
		action_2<<<dimGrid,dimBlock>>>(d_vel_x,d_vel_y,d_den,d_vel_o_x,d_vel_o_y,d_den_o);
	if(counter==144){
		action_3<<<dimGrid,dimBlock>>>(d_vel_x,d_vel_y,d_den,d_vel_o_x,d_vel_o_y,d_den_o);
	}
		
	if(counter==192){
		
		action_4<<<dimGrid,dimBlock>>>(d_vel_x,d_vel_y,d_den,d_vel_o_x,d_vel_o_y,d_den_o);
		
	}
	if(counter==48*5)
		action_5<<<dimGrid,dimBlock>>>(d_vel_x,d_vel_y,d_den,d_vel_o_x,d_vel_o_y,d_den_o);
	if(counter==48*6)
		action_6<<<dimGrid,dimBlock>>>(d_vel_x,d_vel_y,d_den,d_vel_o_x,d_vel_o_y,d_den_o);
	if(counter==48*7)
		action_7<<<dimGrid,dimBlock>>>(d_vel_x,d_vel_y,d_den,d_vel_o_x,d_vel_o_y,d_den_o);

	counter++;
		cout<<"HI";
}

void Lab2VideoGenerator::Generate(uint8_t *yuv) {
	velocity();
	density();
	hipDeviceSynchronize();
	read<<<dimGrid,dimBlock>>>(yuv,d_den,max_den);
	hipDeviceSynchronize();
	hipMemset(yuv+W*H, 128, W*H/2);
	action_h();
	cout<<counter<<" ";
	
}

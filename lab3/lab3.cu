#include "hip/hip_runtime.h"
#include "lab3.h"
#include <cstdio>

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

__global__ void SimpleClone(
	const float *background,
	const float *target,
	const float *mask,
	float *output,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
)
{
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt = wt*yt+xt;
	if (yt < ht and xt < wt and mask[curt] > 127.0f) {
		const int yb = oy+yt, xb = ox+xt;
		const int curb = wb*yb+xb;
		if (0 <= yb and yb < hb and 0 <= xb and xb < wb) {
			output[curb*3+0] = target[curt*3+0];
			output[curb*3+1] = target[curt*3+1];
			output[curb*3+2] = target[curt*3+2];
		}
	}
}


__global__ void CalculateFixed(const float *background,const float* target,const float * mask,float * fixed,const int wb,const int hb,const int wt,const int ht,const int oy,const int ox){
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt = wt*yt+xt;
	int Nt=curt-wt; int Nt_m=Nt; if(yt==0) Nt_m=1; if(yt==0) Nt=curt;
	int St=curt+wt; int St_m=St; if(yt==ht-1) St_m=1; if(yt==ht-1) St=curt;
	int Wt=curt-1;  int Wt_m=Wt; if(xt==0) Wt_m=1; if(xt==0) Wt=curt;
	int Et=curt+1;  int Et_m=Et; if(xt==wt-1) Et_m=1; if(xt==wt-1) Et=curt;

	if (yt < ht and xt < wt and mask[curt] > 127.0f ) {
		const int yb = oy+yt, xb = ox+xt;
		const int curb = wb*yb+xb;
		int Nb=curb-wb;  if(Nb<0) Nb=curb;
		int Sb=curb+wb;  if(Sb>=hb) Sb=curb;
		int Eb=curb+1;   if(Eb>=wb) Eb=curb;
		int Wb=curb-1;   if(Wb<0)  Wb=curb;
		
		if (0 <= yb and yb < hb and 0 <= xb and xb < wb) {
			
			if(mask[Nt_m]> 127.0f and mask[St_m]> 127.0f and mask[Wt_m]> 127.0f and mask[Et_m]> 127.0f){
				fixed[curt*3+0] = 4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0]);
				fixed[curt*3+1] = 4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1]);
				fixed[curt*3+2] = 4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2]);
		
			}
			else if(mask[Nt_m]< 127.0f and mask[St_m]> 127.0f and mask[Wt_m]> 127.0f and mask[Et_m]> 127.0f){
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Nb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Nb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Nb*3+2];
		
			}
			else if(mask[Nt_m]> 127.0f and mask[St_m]< 127.0f and mask[Wt_m]> 127.0f and mask[Et_m]> 127.0f){
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Sb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Sb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Sb*3+2];
			
			}
			else if(mask[Nt_m]> 127.0f and mask[St_m]> 127.0f and mask[Wt_m]< 127.0f and mask[Et_m]> 127.0f){
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Wb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Wb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Wb*3+2];
	
			}
			else if(mask[Nt_m]> 127.0f and mask[St_m]> 127.0f and mask[Wt_m]> 127.0f and mask[Et_m]< 127.0f){
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Eb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Eb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Eb*3+2];
	
			}
			else if(mask[Nt_m]< 127.0f and mask[St_m]< 127.0f and mask[Wt_m]> 127.0f and mask[Et_m]> 127.0f){
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Nb*3+0]+background[Sb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Nb*3+1]+background[Sb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Nb*3+2]+background[Sb*3+2];
		
			}
			else if(mask[Nt_m]< 127.0f and mask[St_m]> 127.0f and mask[Wt_m]< 127.0f and mask[Et_m]> 127.0f){
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Nb*3+0]+background[Wb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Nb*3+1]+background[Wb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Nb*3+2]+background[Wb*3+2];
		
			}
			else if(mask[Nt_m]< 127.0f and mask[St_m]> 127.0f and mask[Wt_m]> 127.0f and mask[Et_m]< 127.0f){
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Nb*3+0]+background[Eb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Nb*3+1]+background[Eb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Nb*3+2]+background[Eb*3+2];
			
			}
			else if(mask[Nt_m]> 127.0f and mask[St_m]< 127.0f and mask[Wt_m]< 127.0f and mask[Et_m]> 127.0f){
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Sb*3+0]+background[Wb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Sb*3+1]+background[Wb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Sb*3+2]+background[Wb*3+2];
	
			}
			else if(mask[Nt_m]> 127.0f and mask[St_m]< 127.0f and mask[Wt_m]> 127.0f and mask[Et_m]< 127.0f){
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Sb*3+0]+background[Eb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Sb*3+1]+background[Eb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Sb*3+2]+background[Eb*3+2];

			}
			else if(mask[Nt_m]> 127.0f and mask[St_m]> 127.0f and mask[Wt_m]< 127.0f and mask[Et_m]< 127.0f){
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Wb*3+0]+background[Eb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Wb*3+1]+background[Eb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Wb*3+2]+background[Eb*3+2];

			}
			else if(mask[Nt_m]< 127.0f and mask[St_m]< 127.0f and mask[Wt_m]< 127.0f and mask[Et_m]> 127.0f){
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Nb*3+0]+background[Sb*3+0]+background[Wb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Nb*3+1]+background[Sb*3+1]+background[Wb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Nb*3+2]+background[Sb*3+2]+background[Wb*3+2];
	
			}
			else if(mask[Nt_m]< 127.0f and mask[St_m]< 127.0f and mask[Wt_m]> 127.0f and mask[Et_m]< 127.0f){
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Nb*3+0]+background[Sb*3+0]+background[Eb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Nb*3+1]+background[Sb*3+1]+background[Eb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Nb*3+2]+background[Sb*3+2]+background[Eb*3+2];

			}
			else if(mask[Nt_m]< 127.0f and mask[St_m]> 127.0f and mask[Wt_m]< 127.0f and mask[Et_m]< 127.0f){
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Nb*3+0]+background[Wb*3+0]+background[Eb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Nb*3+1]+background[Wb*3+1]+background[Eb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Nb*3+2]+background[Wb*3+2]+background[Eb*3+2];

			}
			else if(mask[Nt_m]> 127.0f and mask[St_m]< 127.0f and mask[Wt_m]< 127.0f and mask[Et_m]< 127.0f){
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Sb*3+0]+background[Wb*3+0]+background[Eb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Sb*3+1]+background[Wb*3+1]+background[Eb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Sb*3+2]+background[Wb*3+2]+background[Eb*3+2];

			}
			else{
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Nb*3+0]+background[Wb*3+0]+background[Sb*3+0]+background[Eb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Nb*3+1]+background[Wb*3+1]+background[Sb*3+1]+background[Eb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Nb*3+2]+background[Wb*3+2]+background[Sb*3+2]+background[Eb*3+2];

			}
		}
	}
}


__global__ void PoissonImageCloningIteration(const float *fixed,const float* mask,const float * buf1,float * buf2, const int wt, const int ht){
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt = wt*yt+xt;
		int Nt=curt-wt; int Nt_m=Nt; if(yt==0) Nt_m=1; if(yt==0) Nt=curt;
	int St=curt+wt; int St_m=St; if(yt==ht-1) St_m=1; if(yt==ht-1) St=curt;
	int Wt=curt-1;  int Wt_m=Wt; if(xt==0) Wt_m=1; if(xt==0) Wt=curt;
	int Et=curt+1;  int Et_m=Et; if(xt==wt-1) Et_m=1; if(xt==wt-1) Et=curt;
	if (yt < ht and xt < wt and mask[curt] > 127.0f ) {
		if(mask[Nt_m]> 127.0f and mask[St_m]> 127.0f and mask[Wt_m]> 127.0f and mask[Et_m]> 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(buf1[Nt*3+0]+buf1[St*3+0]+buf1[Wt*3+0]+buf1[Et*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(buf1[Nt*3+1]+buf1[St*3+1]+buf1[Wt*3+1]+buf1[Et*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(buf1[Nt*3+2]+buf1[St*3+2]+buf1[Wt*3+2]+buf1[Et*3+2]))/4;
		}
		else if(mask[Nt_m]< 127.0f and mask[St_m]> 127.0f and mask[Wt_m]> 127.0f and mask[Et_m]> 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(buf1[St*3+0]+buf1[Wt*3+0]+buf1[Et*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(buf1[St*3+1]+buf1[Wt*3+1]+buf1[Et*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(buf1[St*3+2]+buf1[Wt*3+2]+buf1[Et*3+2]))/4;
		}
		else if(mask[Nt_m]> 127.0f and mask[St_m]< 127.0f and mask[Wt_m]> 127.0f and mask[Et_m]> 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(buf1[Nt*3+0]+buf1[Wt*3+0]+buf1[Et*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(buf1[Nt*3+1]+buf1[Wt*3+1]+buf1[Et*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(buf1[Nt*3+2]+buf1[Wt*3+2]+buf1[Et*3+2]))/4;
		}
		else if(mask[Nt_m]> 127.0f and mask[St_m]> 127.0f and mask[Wt_m]< 127.0f and mask[Et_m]> 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(buf1[Nt*3+0]+buf1[St*3+0]+buf1[Et*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(buf1[Nt*3+1]+buf1[St*3+1]+buf1[Et*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(buf1[Nt*3+2]+buf1[St*3+2]+buf1[Et*3+2]))/4;
		}
		else if(mask[Nt_m]> 127.0f and mask[St_m]> 127.0f and mask[Wt_m]> 127.0f and mask[Et_m]< 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(buf1[Nt*3+0]+buf1[St*3+0]+buf1[Wt*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(buf1[Nt*3+1]+buf1[St*3+1]+buf1[Wt*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(buf1[Nt*3+2]+buf1[St*3+2]+buf1[Wt*3+2]))/4;
		}
		else if(mask[Nt_m]< 127.0f and mask[St_m]< 127.0f and mask[Wt_m]> 127.0f and mask[Et_m]> 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(buf1[Wt*3+0]+buf1[Et*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(buf1[Wt*3+1]+buf1[Et*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(buf1[Wt*3+2]+buf1[Et*3+2]))/4;
		}
		else if(mask[Nt_m]< 127.0f and mask[St_m]> 127.0f and mask[Wt_m]< 127.0f and mask[Et_m]> 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(+buf1[St*3+0]+buf1[Et*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(+buf1[St*3+1]+buf1[Et*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(+buf1[St*3+2]+buf1[Et*3+2]))/4;
		}
		else if(mask[Nt_m]< 127.0f and mask[St_m]> 127.0f and mask[Wt_m]> 127.0f and mask[Et_m]< 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(buf1[St*3+0]+buf1[Wt*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(buf1[St*3+1]+buf1[Wt*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(buf1[St*3+2]+buf1[Wt*3+2]))/4;
		}
		else if(mask[Nt_m]> 127.0f and mask[St_m]< 127.0f and mask[Wt_m]< 127.0f and mask[Et_m]> 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(buf1[Nt*3+0]+buf1[Et*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(buf1[Nt*3+1]+buf1[Et*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(buf1[Nt*3+2]+buf1[Et*3+2]))/4;
		}
		else if(mask[Nt_m]> 127.0f and mask[St_m]< 127.0f and mask[Wt_m]> 127.0f and mask[Et_m]< 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(buf1[Nt*3+0]+buf1[Wt*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(buf1[Nt*3+1]+buf1[Wt*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(buf1[Nt*3+2]+buf1[Wt*3+2]))/4;
		}
		else if(mask[Nt_m]> 127.0f and mask[St_m]> 127.0f and mask[Wt_m]< 127.0f and mask[Et_m]< 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(buf1[Nt*3+0]+buf1[St*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(buf1[Nt*3+1]+buf1[St*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(buf1[Nt*3+2]+buf1[St*3+2]))/4;
		}
		else if(mask[Nt_m]< 127.0f and mask[St_m]< 127.0f and mask[Wt_m]< 127.0f and mask[Et_m]> 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(buf1[Et*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(buf1[Et*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(buf1[Et*3+2]))/4;
		}
		else if(mask[Nt_m]< 127.0f and mask[St_m]< 127.0f and mask[Wt_m]> 127.0f and mask[Et_m]< 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(buf1[Wt*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(buf1[Wt*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(buf1[Wt*3+2]))/4;
		}
		else if(mask[Nt_m]< 127.0f and mask[St_m]> 127.0f and mask[Wt_m]< 127.0f and mask[Et_m]< 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(buf1[St*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(buf1[St*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(buf1[St*3+2]))/4;
		}
		else if(mask[Nt_m]> 127.0f and mask[St_m]< 127.0f and mask[Wt_m]< 127.0f and mask[Et_m]< 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(buf1[Nt*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(buf1[Nt*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(buf1[Nt*3+2]))/4;
		}
		else{
			buf2[curt*3+0] = (fixed[curt*3+0])/4;
			buf2[curt*3+1] = (fixed[curt*3+1])/4;
			buf2[curt*3+2] = (fixed[curt*3+2])/4;
		}
	}
}


void PoissonImageCloning(
	const float *background,
	const float *target,
	const float *mask,
	float *output,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
	) {
	// set up
	float *fixed, *buf1, *buf2;
	hipMalloc(&fixed, 3*wt*ht*sizeof(float));
	hipMalloc(&buf1, 3*wt*ht*sizeof(float));
	hipMalloc(&buf2, 3*wt*ht*sizeof(float));
	
	// initialize the iteration
	dim3 gdim(CeilDiv(wt,32), CeilDiv(ht,16)), bdim(32,16);
	
	CalculateFixed<<<gdim, bdim>>>(
	background, target, mask, fixed,
	wb, hb, wt, ht, oy, ox
	);
	hipMemcpy(buf1, target, sizeof(float)*3*wt*ht, hipMemcpyDeviceToDevice);
	
	// iterate
	for (int i = 0; i <20000;i++){
		PoissonImageCloningIteration<<<gdim, bdim>>>(fixed, mask, buf1, buf2, wt, ht);
		PoissonImageCloningIteration<<<gdim, bdim>>>(fixed, mask, buf2, buf1, wt, ht);
	}
	
	// copy the image back
	hipMemcpy(output, background, wb*hb*sizeof(float)*3, hipMemcpyDeviceToDevice);
	
	SimpleClone<<<gdim, bdim>>>(background, buf1, mask, output,wb, hb, wt, ht, oy, ox);
	
	
	
	// clean up
	hipFree(fixed);
	hipFree(buf1);
	hipFree(buf2);
}
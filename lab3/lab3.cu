#include "hip/hip_runtime.h"
#include "lab3.h"
#include <cstdio>

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

__global__ void SimpleClone(
	const float *background,
	const float *target,
	const float *mask,
	float *output,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
)
{
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt = wt*yt+xt;
	if (yt < ht and xt < wt and mask[curt] > 127.0f) {
		const int yb = oy+yt, xb = ox+xt;
		const int curb = wb*yb+xb;
		if (0 <= yb and yb < hb and 0 <= xb and xb < wb) {
			output[curb*3+0] = target[curt*3+0];
			output[curb*3+1] = target[curt*3+1];
			output[curb*3+2] = target[curt*3+2];
		}
	}
}

__global__ void CalculateFixed(const float *background,const float* target,const float * mask,float * fixed,const int wb,const int hb,const int wt,const int ht,const int oy,const int ox){
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt = wt*yt+xt;
	int Nt=curt-wt; int Nt_m=Nt; if(yt==0) Nt_m=0; if(yt==0) Nt=curt;
	int St=curt+wt; int St_m=St; if(yt==ht-1) St_m=0; if(yt==ht-1) St=curt;
	int Wt=curt-1;  int Wt_m=Wt; if(xt==0) Wt_m=0; if(xt==0) Wt=curt;
	int Et=curt+1;  int Et_m=Et; if(xt==wt-1) Et_m=0; if(xt==wt-1) Et=curt;

	if (yt < ht and xt < wt and mask[curt] > 127.0f ) {
		const int yb = oy+yt, xb = ox+xt;
		const int curb = wb*yb+xb;
		int Nb=curb-wb;  if(Nb<0) Nb=curb;
		int Sb=curb+wb;  if(Sb>=hb) Sb=curb;
		int Eb=curb+1;   if(Eb>=wb) Eb=curb;
		int Wb=curb-1;   if(Wb<0)  Wb=curb;
		
		if (0 <= yb and yb < hb and 0 <= xb and xb < wb) {
			
			if(mask[Nt_m]> 127.0f and mask[St_m]> 127.0f and mask[Wt_m]> 127.0f and mask[Et_m]> 127.0f){
				fixed[curt*3+0] = 4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0]);
				fixed[curt*3+1] = 4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1]);
				fixed[curt*3+2] = 4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2]);
		
			}
			else if(mask[Nt_m]< 127.0f and mask[St_m]> 127.0f and mask[Wt_m]> 127.0f and mask[Et_m]> 127.0f){
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Nb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Nb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Nb*3+2];
		
			}
			else if(mask[Nt_m]> 127.0f and mask[St_m]< 127.0f and mask[Wt_m]> 127.0f and mask[Et_m]> 127.0f){
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Sb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Sb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Sb*3+2];
			
			}
			else if(mask[Nt_m]> 127.0f and mask[St_m]> 127.0f and mask[Wt_m]< 127.0f and mask[Et_m]> 127.0f){
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Wb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Wb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Wb*3+2];
	
			}
			else if(mask[Nt_m]> 127.0f and mask[St_m]> 127.0f and mask[Wt_m]> 127.0f and mask[Et_m]< 127.0f){
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Eb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Eb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Eb*3+2];
	
			}
			else if(mask[Nt_m]< 127.0f and mask[St_m]< 127.0f and mask[Wt_m]> 127.0f and mask[Et_m]> 127.0f){
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Nb*3+0]+background[Sb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Nb*3+1]+background[Sb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Nb*3+2]+background[Sb*3+2];
		
			}
			else if(mask[Nt_m]< 127.0f and mask[St_m]> 127.0f and mask[Wt_m]< 127.0f and mask[Et_m]> 127.0f){
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Nb*3+0]+background[Wb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Nb*3+1]+background[Wb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Nb*3+2]+background[Wb*3+2];
		
			}
			else if(mask[Nt_m]< 127.0f and mask[St_m]> 127.0f and mask[Wt_m]> 127.0f and mask[Et_m]< 127.0f){
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Nb*3+0]+background[Eb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Nb*3+1]+background[Eb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Nb*3+2]+background[Eb*3+2];
			
			}
			else if(mask[Nt_m]> 127.0f and mask[St_m]< 127.0f and mask[Wt_m]< 127.0f and mask[Et_m]> 127.0f){
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Sb*3+0]+background[Wb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Sb*3+1]+background[Wb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Sb*3+2]+background[Wb*3+2];
	
			}
			else if(mask[Nt_m]> 127.0f and mask[St_m]< 127.0f and mask[Wt_m]> 127.0f and mask[Et_m]< 127.0f){
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Sb*3+0]+background[Eb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Sb*3+1]+background[Eb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Sb*3+2]+background[Eb*3+2];

			}
			else if(mask[Nt_m]> 127.0f and mask[St_m]> 127.0f and mask[Wt_m]< 127.0f and mask[Et_m]< 127.0f){
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Wb*3+0]+background[Eb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Wb*3+1]+background[Eb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Wb*3+2]+background[Eb*3+2];

			}
			else if(mask[Nt_m]< 127.0f and mask[St_m]< 127.0f and mask[Wt_m]< 127.0f and mask[Et_m]> 127.0f){
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Nb*3+0]+background[Sb*3+0]+background[Wb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Nb*3+1]+background[Sb*3+1]+background[Wb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Nb*3+2]+background[Sb*3+2]+background[Wb*3+2];
	
			}
			else if(mask[Nt_m]< 127.0f and mask[St_m]< 127.0f and mask[Wt_m]> 127.0f and mask[Et_m]< 127.0f){
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Nb*3+0]+background[Sb*3+0]+background[Eb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Nb*3+1]+background[Sb*3+1]+background[Eb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Nb*3+2]+background[Sb*3+2]+background[Eb*3+2];

			}
			else if(mask[Nt_m]< 127.0f and mask[St_m]> 127.0f and mask[Wt_m]< 127.0f and mask[Et_m]< 127.0f){
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Nb*3+0]+background[Wb*3+0]+background[Eb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Nb*3+1]+background[Wb*3+1]+background[Eb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Nb*3+2]+background[Wb*3+2]+background[Eb*3+2];

			}
			else if(mask[Nt_m]> 127.0f and mask[St_m]< 127.0f and mask[Wt_m]< 127.0f and mask[Et_m]< 127.0f){
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Sb*3+0]+background[Wb*3+0]+background[Eb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Sb*3+1]+background[Wb*3+1]+background[Eb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Sb*3+2]+background[Wb*3+2]+background[Eb*3+2];

			}
			else{
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Nb*3+0]+background[Wb*3+0]+background[Sb*3+0]+background[Eb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Nb*3+1]+background[Wb*3+1]+background[Sb*3+1]+background[Eb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Nb*3+2]+background[Wb*3+2]+background[Sb*3+2]+background[Eb*3+2];

			}
		}
	}
}


__global__ void PoissonImageCloningIteration(const float *fixed,const float* mask,const float * buf1,float * buf2, const int wt, const int ht){
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt = wt*yt+xt;
		int Nt=curt-wt; int Nt_m=Nt; if(yt==0) Nt_m=0; if(yt==0) Nt=curt;
	int St=curt+wt; int St_m=St; if(yt==ht-1) St_m=0; if(yt==ht-1) St=curt;
	int Wt=curt-1;  int Wt_m=Wt; if(xt==0) Wt_m=0; if(xt==0) Wt=curt;
	int Et=curt+1;  int Et_m=Et; if(xt==wt-1) Et_m=0; if(xt==wt-1) Et=curt;
	if (yt < ht and xt < wt and mask[curt] > 127.0f ) {
		if(mask[Nt_m]> 127.0f and mask[St_m]> 127.0f and mask[Wt_m]> 127.0f and mask[Et_m]> 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(buf1[Nt*3+0]+buf1[St*3+0]+buf1[Wt*3+0]+buf1[Et*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(buf1[Nt*3+1]+buf1[St*3+1]+buf1[Wt*3+1]+buf1[Et*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(buf1[Nt*3+2]+buf1[St*3+2]+buf1[Wt*3+2]+buf1[Et*3+2]))/4;
		}
		else if(mask[Nt_m]< 127.0f and mask[St_m]> 127.0f and mask[Wt_m]> 127.0f and mask[Et_m]> 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(buf1[St*3+0]+buf1[Wt*3+0]+buf1[Et*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(buf1[St*3+1]+buf1[Wt*3+1]+buf1[Et*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(buf1[St*3+2]+buf1[Wt*3+2]+buf1[Et*3+2]))/4;
		}
		else if(mask[Nt_m]> 127.0f and mask[St_m]< 127.0f and mask[Wt_m]> 127.0f and mask[Et_m]> 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(buf1[Nt*3+0]+buf1[Wt*3+0]+buf1[Et*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(buf1[Nt*3+1]+buf1[Wt*3+1]+buf1[Et*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(buf1[Nt*3+2]+buf1[Wt*3+2]+buf1[Et*3+2]))/4;
		}
		else if(mask[Nt_m]> 127.0f and mask[St_m]> 127.0f and mask[Wt_m]< 127.0f and mask[Et_m]> 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(buf1[Nt*3+0]+buf1[St*3+0]+buf1[Et*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(buf1[Nt*3+1]+buf1[St*3+1]+buf1[Et*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(buf1[Nt*3+2]+buf1[St*3+2]+buf1[Et*3+2]))/4;
		}
		else if(mask[Nt_m]> 127.0f and mask[St_m]> 127.0f and mask[Wt_m]> 127.0f and mask[Et_m]< 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(buf1[Nt*3+0]+buf1[St*3+0]+buf1[Wt*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(buf1[Nt*3+1]+buf1[St*3+1]+buf1[Wt*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(buf1[Nt*3+2]+buf1[St*3+2]+buf1[Wt*3+2]))/4;
		}
		else if(mask[Nt_m]< 127.0f and mask[St_m]< 127.0f and mask[Wt_m]> 127.0f and mask[Et_m]> 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(buf1[Wt*3+0]+buf1[Et*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(buf1[Wt*3+1]+buf1[Et*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(buf1[Wt*3+2]+buf1[Et*3+2]))/4;
		}
		else if(mask[Nt_m]< 127.0f and mask[St_m]> 127.0f and mask[Wt_m]< 127.0f and mask[Et_m]> 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(+buf1[St*3+0]+buf1[Et*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(+buf1[St*3+1]+buf1[Et*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(+buf1[St*3+2]+buf1[Et*3+2]))/4;
		}
		else if(mask[Nt_m]< 127.0f and mask[St_m]> 127.0f and mask[Wt_m]> 127.0f and mask[Et_m]< 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(buf1[St*3+0]+buf1[Wt*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(buf1[St*3+1]+buf1[Wt*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(buf1[St*3+2]+buf1[Wt*3+2]))/4;
		}
		else if(mask[Nt_m]> 127.0f and mask[St_m]< 127.0f and mask[Wt_m]< 127.0f and mask[Et_m]> 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(buf1[Nt*3+0]+buf1[Et*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(buf1[Nt*3+1]+buf1[Et*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(buf1[Nt*3+2]+buf1[Et*3+2]))/4;
		}
		else if(mask[Nt_m]> 127.0f and mask[St_m]< 127.0f and mask[Wt_m]> 127.0f and mask[Et_m]< 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(buf1[Nt*3+0]+buf1[Wt*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(buf1[Nt*3+1]+buf1[Wt*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(buf1[Nt*3+2]+buf1[Wt*3+2]))/4;
		}
		else if(mask[Nt_m]> 127.0f and mask[St_m]> 127.0f and mask[Wt_m]< 127.0f and mask[Et_m]< 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(buf1[Nt*3+0]+buf1[St*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(buf1[Nt*3+1]+buf1[St*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(buf1[Nt*3+2]+buf1[St*3+2]))/4;
		}
		else if(mask[Nt_m]< 127.0f and mask[St_m]< 127.0f and mask[Wt_m]< 127.0f and mask[Et_m]> 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(buf1[Et*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(buf1[Et*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(buf1[Et*3+2]))/4;
		}
		else if(mask[Nt_m]< 127.0f and mask[St_m]< 127.0f and mask[Wt_m]> 127.0f and mask[Et_m]< 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(buf1[Wt*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(buf1[Wt*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(buf1[Wt*3+2]))/4;
		}
		else if(mask[Nt_m]< 127.0f and mask[St_m]> 127.0f and mask[Wt_m]< 127.0f and mask[Et_m]< 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(buf1[St*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(buf1[St*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(buf1[St*3+2]))/4;
		}
		else if(mask[Nt_m]> 127.0f and mask[St_m]< 127.0f and mask[Wt_m]< 127.0f and mask[Et_m]< 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(buf1[Nt*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(buf1[Nt*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(buf1[Nt*3+2]))/4;
		}
		else{
			buf2[curt*3+0] = (fixed[curt*3+0])/4;
			buf2[curt*3+1] = (fixed[curt*3+1])/4;
			buf2[curt*3+2] = (fixed[curt*3+2])/4;
		}
	}
}




__global__ void shrink_a(const float *target,float *target_a,const float *mask,float *mask_a,const int wt,const int ht,const int wt_a,const int ht_a){
	const int yt_a = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt_a = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt_a = wt_a*yt_a+xt_a;
	if (yt_a < ht_a and xt_a < wt_a) {
		for(int row=0;row<8;row++){
			for(int col=0;col<8;col++){
				target_a[curt_a*3+0]+=target[(wt*(yt_a*8+row)+(xt_a*8+col))*3+0];
				target_a[curt_a*3+1]+=target[(wt*(yt_a*8+row)+(xt_a*8+col))*3+1];
				target_a[curt_a*3+2]+=target[(wt*(yt_a*8+row)+(xt_a*8+col))*3+2];
				mask_a[curt_a*3+0]+=mask[(wt*(yt_a*8+row)+(xt_a*8+col))*3+0];
				mask_a[curt_a*3+1]+=mask[(wt*(yt_a*8+row)+(xt_a*8+col))*3+1];
				mask_a[curt_a*3+2]+=mask[(wt*(yt_a*8+row)+(xt_a*8+col))*3+2];
			}
		}
		target_a[curt_a*3+0]/=64;
		target_a[curt_a*3+1]/=64;
		target_a[curt_a*3+2]/=64;
		mask_a[curt_a*3+0]/=64;
		mask_a[curt_a*3+1]/=64;
		mask_a[curt_a*3+2]/=64;
	}
}

__global__ void shrink_b(const float *target,float *target_a,const float *mask,float *mask_a,const int wt,const int ht,const int wt_a,const int ht_a){
	const int yt_a = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt_a = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt_a = wt_a*yt_a+xt_a;
	if (yt_a < ht_a and xt_a < wt_a) {
		for(int row=0;row<4;row++){
			for(int col=0;col<4;col++){
				target_a[curt_a*3+0]+=target[(wt*(yt_a*4+row)+(xt_a*4+col))*3+0];
				target_a[curt_a*3+1]+=target[(wt*(yt_a*4+row)+(xt_a*4+col))*3+1];
				target_a[curt_a*3+2]+=target[(wt*(yt_a*4+row)+(xt_a*4+col))*3+2];
				mask_a[curt_a*3+0]+=	mask[(wt*(yt_a*4+row)+(xt_a*4+col))*3+0];
				mask_a[curt_a*3+1]+=	mask[(wt*(yt_a*4+row)+(xt_a*4+col))*3+1];
				mask_a[curt_a*3+2]+=	mask[(wt*(yt_a*4+row)+(xt_a*4+col))*3+2];
			}
		}
		target_a[curt_a*3+0]/=16;
		target_a[curt_a*3+1]/=16;
		target_a[curt_a*3+2]/=16;
		  mask_a[curt_a*3+0]/=16;
		  mask_a[curt_a*3+1]/=16;
		  mask_a[curt_a*3+2]/=16;
	}
}

__global__ void shrink_c(const float *target,float *target_a,const float *mask,float *mask_a,const int wt,const int ht,const int wt_a,const int ht_a){
	const int yt_a = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt_a = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt_a = wt_a*yt_a+xt_a;
	if (yt_a < ht_a and xt_a < wt_a) {
		for(int row=0;row<2;row++){
			for(int col=0;col<2;col++){
				target_a[curt_a*3+0]+=target[(wt*(yt_a*2+row)+(xt_a*2+col))*3+0];
				target_a[curt_a*3+1]+=target[(wt*(yt_a*2+row)+(xt_a*2+col))*3+1];
				target_a[curt_a*3+2]+=target[(wt*(yt_a*2+row)+(xt_a*2+col))*3+2];
				mask_a[curt_a*3+0]+=	mask[(wt*(yt_a*2+row)+(xt_a*2+col))*3+0];
				mask_a[curt_a*3+1]+=	mask[(wt*(yt_a*2+row)+(xt_a*2+col))*3+1];
				mask_a[curt_a*3+2]+=	mask[(wt*(yt_a*2+row)+(xt_a*2+col))*3+2];
			}
		}
		target_a[curt_a*3+0]/=4;
		target_a[curt_a*3+1]/=4;
		target_a[curt_a*3+2]/=4;
		  mask_a[curt_a*3+0]/=4;
		  mask_a[curt_a*3+1]/=4;
		  mask_a[curt_a*3+2]/=4;
	}
}

__global__ void enlarge_a(const float *buf1_a, float *output_a,const int wt,const int ht,const int wt_a,const int ht_a){
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt = wt*yt+xt;
	if (yt < ht and xt < wt) {
		output_a[curt*3+0]=buf1_a[(wt_a*(yt/2)+xt/2)*3+0];
		output_a[curt*3+1]=buf1_a[(wt_a*(yt/2)+xt/2)*3+1];
		output_a[curt*3+2]=buf1_a[(wt_a*(yt/2)+xt/2)*3+2];
	}
}

__global__ void enlarge_b(const float *buf1_a, float *output_a,const int wt,const int ht,const int wt_a,const int ht_a){
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt = wt*yt+xt;
	if (yt < ht and xt < wt) {
		output_a[curt*3+0]=buf1_a[(wt_a*(yt/2)+xt/2)*3+0];
		output_a[curt*3+1]=buf1_a[(wt_a*(yt/2)+xt/2)*3+1];
		output_a[curt*3+2]=buf1_a[(wt_a*(yt/2)+xt/2)*3+2];
	}
}

__global__ void enlarge_c(const float *buf1_a, float *output_a,const int wt,const int ht,const int wt_a,const int ht_a){
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt = wt*yt+xt;
	if (yt < ht and xt < wt) {
		output_a[curt*3+0]=buf1_a[(wt_a*(yt/2)+xt/2)*3+0];
		output_a[curt*3+1]=buf1_a[(wt_a*(yt/2)+xt/2)*3+1];
		output_a[curt*3+2]=buf1_a[(wt_a*(yt/2)+xt/2)*3+2];
	}
}

__global__ void shrinkbackground_a(const float *background,float *background_a,const int wb,const int hb,const int wb_a,const int hb_a){
	const int yb_a = blockIdx.y * blockDim.y + threadIdx.y;
	const int xb_a = blockIdx.x * blockDim.x + threadIdx.x;
	const int curb_a = wb_a*yb_a+xb_a;
	if (yb_a < hb_a and xb_a < wb_a) {
		for(int row=0;row<8;row++){
			for(int col=0;col<8;col++){
				background_a[curb_a*3+0]+=background[(wb*(yb_a*8+row)+(xb_a*8+col))*3+0];
				background_a[curb_a*3+1]+=background[(wb*(yb_a*8+row)+(xb_a*8+col))*3+1];
				background_a[curb_a*3+2]+=background[(wb*(yb_a*8+row)+(xb_a*8+col))*3+2];
			}
		}
		background_a[curb_a*3+0]/=64;
		background_a[curb_a*3+1]/=64;
		background_a[curb_a*3+2]/=64;
	}
}

__global__ void shrinkbackground_b(const float *background,float *background_a,const int wb,const int hb,const int wb_a,const int hb_a){
	const int yb_a = blockIdx.y * blockDim.y + threadIdx.y;
	const int xb_a = blockIdx.x * blockDim.x + threadIdx.x;
	const int curb_a = wb_a*yb_a+xb_a;
	if (yb_a < hb_a and xb_a < wb_a) {
		for(int row=0;row<4;row++){
			for(int col=0;col<4;col++){
				background_a[curb_a*3+0]+=background[(wb*(yb_a*4+row)+(xb_a*4+col))*3+0];
				background_a[curb_a*3+1]+=background[(wb*(yb_a*4+row)+(xb_a*4+col))*3+1];
				background_a[curb_a*3+2]+=background[(wb*(yb_a*4+row)+(xb_a*4+col))*3+2];
			}
		}
		background_a[curb_a*3+0]/=16;
		background_a[curb_a*3+1]/=16;
		background_a[curb_a*3+2]/=16;
	}
}

__global__ void shrinkbackground_c(const float *background,float *background_a,const int wb,const int hb,const int wb_a,const int hb_a){
	const int yb_a = blockIdx.y * blockDim.y + threadIdx.y;
	const int xb_a = blockIdx.x * blockDim.x + threadIdx.x;
	const int curb_a = wb_a*yb_a+xb_a;
	if (yb_a < hb_a and xb_a < wb_a) {
		for(int row=0;row<2;row++){
			for(int col=0;col<2;col++){
				background_a[curb_a*3+0]+=background[(wb*(yb_a*2+row)+(xb_a*2+col))*3+0];
				background_a[curb_a*3+1]+=background[(wb*(yb_a*2+row)+(xb_a*2+col))*3+1];
				background_a[curb_a*3+2]+=background[(wb*(yb_a*2+row)+(xb_a*2+col))*3+2];
			}
		}
		background_a[curb_a*3+0]/=4;
		background_a[curb_a*3+1]/=4;
		background_a[curb_a*3+2]/=4;
	}
}

void PoissonImageCloning(
	const float *background,
	const float *target,
	const float *mask,
	float *output,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
	) {
		
	int wt_a=wt/8;
	int ht_a=ht/8;
	int wb_a=wb/8;
	int hb_a=hb/8;
	int oy_a=oy/8;
	int ox_a=ox/8;
	int wt_b=wt/4;
	int ht_b=ht/4;
	int wb_b=wb/4;
	int hb_b=hb/4;
	int oy_b=oy/4;
	int ox_b=ox/4;
	int wt_c=wt/2;
	int ht_c=ht/2;
	int wb_c=wb/2;
	int hb_c=hb/2;
	int oy_c=oy/2;
	int ox_c=ox/2;
	// set up
	float *fixed, *buf1, *buf2;
	float *fixed_a, *buf1_a, *buf2_a, *background_a,*output_a,*mask_a,*target_a;
	float *fixed_b, *buf1_b, *buf2_b, *background_b,*output_b,*mask_b,*target_b;
	float *fixed_c, *buf1_c, *buf2_c, *background_c,*output_c,*mask_c,*target_c;
	hipMalloc(&fixed, 3*wt*ht*sizeof(float));
	hipMalloc(&buf1, 3*wt*ht*sizeof(float));
	hipMalloc(&buf2, 3*wt*ht*sizeof(float));
	hipMalloc(&fixed_a, 		3*wt_a*ht_a*sizeof(float));
	hipMalloc(&buf1_a,  		3*wt_a*ht_a*sizeof(float));
	hipMalloc(&output_a,  		3*wt_b*ht_b*sizeof(float));
	hipMalloc(&buf2_a,  		3*wt_a*ht_a*sizeof(float));
	hipMalloc(&background_a, 	3*wb_a*hb_a*sizeof(float));
	hipMalloc(&target_a, 		3*wt_a*ht_a*sizeof(float));
	hipMalloc(&mask_a, 		3*wt_a*ht_a*sizeof(float));
	hipMalloc(&fixed_b, 		3*wt_b*ht_b*sizeof(float));
	hipMalloc(&buf1_b,  		3*wt_b*ht_b*sizeof(float));
	hipMalloc(&output_b,  		3*wt_c*ht_c*sizeof(float));
	hipMalloc(&buf2_b,  		3*wt_b*ht_b*sizeof(float));
	hipMalloc(&background_b, 	3*wb_b*hb_b*sizeof(float));
	hipMalloc(&target_b, 		3*wt_b*ht_b*sizeof(float));
	hipMalloc(&mask_b, 		3*wt_b*ht_b*sizeof(float));
	hipMalloc(&fixed_c, 		3*wt_c*ht_c*sizeof(float));
	hipMalloc(&buf1_c,  		3*wt_c*ht_c*sizeof(float));
	hipMalloc(&output_c,  		3*wt*ht*sizeof(float));
	hipMalloc(&buf2_c,  		3*wt_c*ht_c*sizeof(float));
	hipMalloc(&background_c, 	3*wb_c*hb_c*sizeof(float));
	hipMalloc(&target_c, 		3*wt_c*ht_c*sizeof(float));
	hipMalloc(&mask_c, 		3*wt_c*ht_c*sizeof(float));
	
	dim3 gdim_a(CeilDiv(wt_a,32), CeilDiv(ht_a,16)), bdim_a(32,16);
	dim3 gdim_back_a(CeilDiv(wb_a,32), CeilDiv(hb_a,16)), bdim_back_a(32,16);
	dim3 gdim_b(CeilDiv(wt_b,32), CeilDiv(ht_b,16)), bdim_b(32,16);
	dim3 gdim_back_b(CeilDiv(wb_b,32), CeilDiv(hb_b,16)), bdim_back_b(32,16);
	dim3 gdim_c(CeilDiv(wt_c,32), CeilDiv(ht_c,16)), bdim_c(32,16);
	dim3 gdim_back_c(CeilDiv(wb_c,32), CeilDiv(hb_c,16)), bdim_back_c(32,16);
	// initialize the iteration
	dim3 gdim(CeilDiv(wt,32), CeilDiv(ht,16)), bdim(32,16);
	CalculateFixed<<<gdim, bdim>>>(
	background, target, mask, fixed,
	wb, hb, wt, ht, oy, ox
	);
//====================================	
	
	
	shrink_a<<<gdim_a,bdim_a>>>(target,target_a,mask,mask_a,wt,ht,wt_a,ht_a);
	shrinkbackground_a<<<gdim_back_a,bdim_back_a>>>(background,background_a,wb,hb,wb_a,hb_a);
	// initialize the iteration
	
	CalculateFixed
	<<<gdim_a, bdim_a>>>(
	background_a, target_a, mask_a, fixed_a,
	wb_a, hb_a, wt_a, ht_a, oy_a, ox_a
	);
	hipMemcpy(buf1_a, target_a, sizeof(float)*3*wt_a*ht_a, hipMemcpyDeviceToDevice);
	
	// iterate
	for (int i = 0; i <1000;i++){
		PoissonImageCloningIteration<<<gdim_a, bdim_a>>>(fixed_a, mask_a, buf1_a, buf2_a, wt_a, ht_a);
		PoissonImageCloningIteration<<<gdim_a, bdim_a>>>(fixed_a, mask_a, buf2_a, buf1_a, wt_a, ht_a);
	}
	enlarge_a<<<gdim_b,bdim_b>>>(buf1_a,output_a,wt_b,ht_b,wt_a,ht_a);
//================================_
	
	shrink_b<<<gdim_b,bdim_b>>>(target,target_b,mask,mask_b,wt,ht,wt_b,ht_b);
	shrinkbackground_b<<<gdim_back_b,bdim_back_b>>>(background,background_b,wb,hb,wb_b,hb_b);
	// initialize the iteration
	
	CalculateFixed
	<<<gdim_b, bdim_b>>>(
	background_b, target_b, mask_b, fixed_b,
	wb_b, hb_b, wt_b, ht_b, oy_b, ox_b
	);
	hipMemcpy(buf1_b, output_a, sizeof(float)*3*wt_b*ht_b, hipMemcpyDeviceToDevice);
	
	// iterate
	for (int i = 0; i <1000;i++){
		PoissonImageCloningIteration<<<gdim_b, bdim_b>>>(fixed_b, mask_b, buf1_b, buf2_b, wt_b, ht_b);
		PoissonImageCloningIteration<<<gdim_b, bdim_b>>>(fixed_b, mask_b, buf2_b, buf1_b, wt_b, ht_b);
	}
	enlarge_b<<<gdim_c,bdim_c>>>(buf1_b,output_b,wt_c,ht_c,wt_b,ht_b);
	

//=================================
	
	shrink_c<<<gdim_c,bdim_c>>>(target,target_c,mask,mask_c,wt,ht,wt_c,ht_c);
	shrinkbackground_c<<<gdim_back_c,bdim_back_c>>>(background,background_c,wb,hb,wb_c,hb_c);
	// initialize the iteration
	
	CalculateFixed
	<<<gdim_c, bdim_c>>>(
	background_c, target_c, mask_c, fixed_c,
	wb_c, hb_c, wt_c, ht_c, oy_c, ox_c
	);
	hipMemcpy(buf1_c, output_b, sizeof(float)*3*wt_c*ht_c, hipMemcpyDeviceToDevice);
	
	// iterate
	for (int i = 0; i <1000;i++){
		PoissonImageCloningIteration<<<gdim_c, bdim_c>>>(fixed_c, mask_c, buf1_c, buf2_c, wt_c, ht_c);
		PoissonImageCloningIteration<<<gdim_c, bdim_c>>>(fixed_c, mask_c, buf2_c, buf1_c, wt_c, ht_c);
	}
	enlarge_c<<<gdim,bdim>>>(buf1_c,output_c,wt,ht,wt_c,ht_c);
	
	
//=================================
	hipMemset(buf1,0,sizeof(float)*3*wt*ht);
	hipMemcpy(buf1, output_c, sizeof(float)*3*wt*ht, hipMemcpyDeviceToDevice);
	for (int i = 0; i <1000;i++){
		PoissonImageCloningIteration<<<gdim, bdim>>>(fixed, mask, buf1, buf2, wt, ht);
		PoissonImageCloningIteration<<<gdim, bdim>>>(fixed, mask, buf2, buf1, wt, ht);
	}
	
	// copy the image back
	hipMemcpy(output, background, wb*hb*sizeof(float)*3, hipMemcpyDeviceToDevice);
	
	SimpleClone<<<gdim, bdim>>>(background, buf1, mask, output,wb, hb, wt, ht, oy, ox);
	
	
	
	// clean up
	hipFree(fixed);
	hipFree(buf1);
	hipFree(buf2);
	hipFree(fixed_a);
	hipFree(buf1_a);
	hipFree(buf2_a);
	hipFree(output_a);
	hipFree(background_a);
	hipFree(target_a);
	hipFree(mask_a);
	
	hipFree(fixed_b);
	hipFree(buf1_b);
	hipFree(buf2_b);
	hipFree(output_b);
	hipFree(background_b);
	hipFree(target_b);
	hipFree(mask_b);
	
	hipFree(fixed_c);
	hipFree(buf1_c);
	hipFree(buf2_c);
	hipFree(output_c);
	hipFree(background_c);
	hipFree(target_c);
	hipFree(mask_c);
//=======================================	
	
	
	
	
	
	
	
}
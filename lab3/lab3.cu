#include "hip/hip_runtime.h"
#include "lab3.h"
#include <cstdio>

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

__global__ void SimpleClone(
	const float *background,
	const float *target,
	const float *mask,
	float *output,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
)
{
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt = wt*yt+xt;
	if (yt < ht and xt < wt and mask[curt] > 127.0f) {
		const int yb = oy+yt, xb = ox+xt;
		const int curb = wb*yb+xb;
		if (0 <= yb and yb < hb and 0 <= xb and xb < wb) {
			output[curb*3+0] = target[curt*3+0];
			output[curb*3+1] = target[curt*3+1];
			output[curb*3+2] = target[curt*3+2];
		}
	}
}


__global__ void CalculateFixed(const float *background,const float* target,const float * mask,float * fixed,const int wb,const int hb,const int wt,const int ht,const int oy,const int ox){
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt = wt*yt+xt;
	int Nt=curt-wt;
	int St=curt+wt;
	int Wt=curt-1;
	int Et=curt+1;
	if (yt < ht and xt < wt and mask[curt] > 127.0f and yt>1 and xt>1) {
		const int yb = oy+yt, xb = ox+xt;
		const int curb = wb*yb+xb;
		int Nb=curb-wb;
		int Sb=curb+wb;
		int Eb=curb+1;
		int Wb=curb+1;
		if (1 <= yb and yb < hb-1 and 1 <= xb and xb < wb-1) {
			if(mask[Nt]> 127.0f and mask[St]> 127.0f and mask[Wt]> 127.0f and mask[Et]> 127.0f){
				fixed[curt*3+0] = 4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0]);
				fixed[curt*3+1] = 4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1]);
				fixed[curt*3+2] = 4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2]);
				
			}
			else if(mask[Nt]< 127.0f and mask[St]> 127.0f and mask[Wt]> 127.0f and mask[Et]> 127.0f){
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Nb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Nb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Nb*3+2];
			}
			else if(mask[Nt]> 127.0f and mask[St]< 127.0f and mask[Wt]> 127.0f and mask[Et]> 127.0f){
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Sb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Sb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Sb*3+2];
			}
			else if(mask[Nt]> 127.0f and mask[St]> 127.0f and mask[Wt]< 127.0f and mask[Et]> 127.0f){
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Wb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Wb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Wb*3+2];
			}
			else if(mask[Nt]> 127.0f and mask[St]> 127.0f and mask[Wt]> 127.0f and mask[Et]< 127.0f){
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Eb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Eb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Eb*3+2];
			}
			else if(mask[Nt]< 127.0f and mask[St]< 127.0f and mask[Wt]> 127.0f and mask[Et]> 127.0f){
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Nb*3+0]+background[Sb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Nb*3+1]+background[Sb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Nb*3+2]+background[Sb*3+2];
			}
			else if(mask[Nt]< 127.0f and mask[St]> 127.0f and mask[Wt]< 127.0f and mask[Et]> 127.0f){
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Nb*3+0]+background[Wb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Nb*3+1]+background[Wb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Nb*3+2]+background[Wb*3+2];
			}
			else if(mask[Nt]< 127.0f and mask[St]> 127.0f and mask[Wt]> 127.0f and mask[Et]< 127.0f){
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Nb*3+0]+background[Eb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Nb*3+1]+background[Eb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Nb*3+2]+background[Eb*3+2];
			}
			else if(mask[Nt]> 127.0f and mask[St]< 127.0f and mask[Wt]< 127.0f and mask[Et]> 127.0f){
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Sb*3+0]+background[Wb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Sb*3+1]+background[Wb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Sb*3+2]+background[Wb*3+2];
			}
			else if(mask[Nt]> 127.0f and mask[St]< 127.0f and mask[Wt]> 127.0f and mask[Et]< 127.0f){
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Sb*3+0]+background[Eb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Sb*3+1]+background[Eb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Sb*3+2]+background[Eb*3+2];
			}
			else if(mask[Nt]> 127.0f and mask[St]> 127.0f and mask[Wt]< 127.0f and mask[Et]< 127.0f){
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Wb*3+0]+background[Eb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Wb*3+1]+background[Eb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Wb*3+2]+background[Eb*3+2];
			}
			else if(mask[Nt]< 127.0f and mask[St]< 127.0f and mask[Wt]< 127.0f and mask[Et]> 127.0f){
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Nb*3+0]+background[Sb*3+0]+background[Wb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Nb*3+1]+background[Sb*3+1]+background[Wb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Nb*3+2]+background[Sb*3+2]+background[Wb*3+2];
			}
			else if(mask[Nt]< 127.0f and mask[St]< 127.0f and mask[Wt]> 127.0f and mask[Et]< 127.0f){
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Nb*3+0]+background[Sb*3+0]+background[Eb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Nb*3+1]+background[Sb*3+1]+background[Eb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Nb*3+2]+background[Sb*3+2]+background[Eb*3+2];
			}
			else if(mask[Nt]< 127.0f and mask[St]> 127.0f and mask[Wt]< 127.0f and mask[Et]< 127.0f){
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Nb*3+0]+background[Wb*3+0]+background[Eb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Nb*3+1]+background[Wb*3+1]+background[Eb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Nb*3+2]+background[Wb*3+2]+background[Eb*3+2];
			}
			else if(mask[Nt]> 127.0f and mask[St]< 127.0f and mask[Wt]< 127.0f and mask[Et]< 127.0f){
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Sb*3+0]+background[Wb*3+0]+background[Eb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Sb*3+1]+background[Wb*3+1]+background[Eb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Sb*3+2]+background[Wb*3+2]+background[Eb*3+2];
			}
			else{
				fixed[curt*3+0] =4*target[curt*3+0]-(target[Nt*3+0]+target[St*3+0]+target[Wt*3+0]+target[Et*3+0])+background[Nb*3+0]+background[Wb*3+0]+background[Sb*3+0]+background[Eb*3+0];
				fixed[curt*3+1] =4*target[curt*3+1]-(target[Nt*3+1]+target[St*3+1]+target[Wt*3+1]+target[Et*3+1])+background[Nb*3+1]+background[Wb*3+1]+background[Sb*3+1]+background[Eb*3+1];
				fixed[curt*3+2] =4*target[curt*3+2]-(target[Nt*3+2]+target[St*3+2]+target[Wt*3+2]+target[Et*3+2])+background[Nb*3+2]+background[Wb*3+2]+background[Sb*3+2]+background[Eb*3+2];
			}
		}
	}
}


__global__ void PoissonImageCloningIteration(const float *fixed,const float* mask,const float * buf1,float * buf2, const int wt, const int ht){
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt = wt*yt+xt;
	int Nt=curt-wt;
	int St=curt+wt;
	int Wt=curt-1;
	int Et=curt+1;
	if (yt < ht and xt < wt and mask[curt] > 127.0f and yt>1 and xt>1) {
		if(mask[Nt]> 127.0f and mask[St]> 127.0f and mask[Wt]> 127.0f and mask[Et]> 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(buf1[Nt*3+0]+buf1[St*3+0]+buf1[Wt*3+0]+buf1[Et*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(buf1[Nt*3+1]+buf1[St*3+1]+buf1[Wt*3+1]+buf1[Et*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(buf1[Nt*3+2]+buf1[St*3+2]+buf1[Wt*3+2]+buf1[Et*3+2]))/4;
		}
		else if(mask[Nt]< 127.0f and mask[St]> 127.0f and mask[Wt]> 127.0f and mask[Et]> 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(buf1[St*3+0]+buf1[Wt*3+0]+buf1[Et*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(buf1[St*3+1]+buf1[Wt*3+1]+buf1[Et*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(buf1[St*3+2]+buf1[Wt*3+2]+buf1[Et*3+2]))/4;
		}
		else if(mask[Nt]> 127.0f and mask[St]< 127.0f and mask[Wt]> 127.0f and mask[Et]> 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(buf1[Nt*3+0]+buf1[Wt*3+0]+buf1[Et*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(buf1[Nt*3+1]+buf1[Wt*3+1]+buf1[Et*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(buf1[Nt*3+2]+buf1[Wt*3+2]+buf1[Et*3+2]))/4;
		}
		else if(mask[Nt]> 127.0f and mask[St]> 127.0f and mask[Wt]< 127.0f and mask[Et]> 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(buf1[Nt*3+0]+buf1[St*3+0]+buf1[Et*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(buf1[Nt*3+1]+buf1[St*3+1]+buf1[Et*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(buf1[Nt*3+2]+buf1[St*3+2]+buf1[Et*3+2]))/4;
		}
		else if(mask[Nt]> 127.0f and mask[St]> 127.0f and mask[Wt]> 127.0f and mask[Et]< 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(buf1[Nt*3+0]+buf1[St*3+0]+buf1[Wt*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(buf1[Nt*3+1]+buf1[St*3+1]+buf1[Wt*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(buf1[Nt*3+2]+buf1[St*3+2]+buf1[Wt*3+2]))/4;
		}
		else if(mask[Nt]< 127.0f and mask[St]< 127.0f and mask[Wt]> 127.0f and mask[Et]> 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(buf1[Wt*3+0]+buf1[Et*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(buf1[Wt*3+1]+buf1[Et*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(buf1[Wt*3+2]+buf1[Et*3+2]))/4;
		}
		else if(mask[Nt]< 127.0f and mask[St]> 127.0f and mask[Wt]< 127.0f and mask[Et]> 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(+buf1[St*3+0]+buf1[Et*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(+buf1[St*3+1]+buf1[Et*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(+buf1[St*3+2]+buf1[Et*3+2]))/4;
		}
		else if(mask[Nt]< 127.0f and mask[St]> 127.0f and mask[Wt]> 127.0f and mask[Et]< 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(buf1[St*3+0]+buf1[Wt*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(buf1[St*3+1]+buf1[Wt*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(buf1[St*3+2]+buf1[Wt*3+2]))/4;
		}
		else if(mask[Nt]> 127.0f and mask[St]< 127.0f and mask[Wt]< 127.0f and mask[Et]> 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(buf1[Nt*3+0]+buf1[Et*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(buf1[Nt*3+1]+buf1[Et*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(buf1[Nt*3+2]+buf1[Et*3+2]))/4;
		}
		else if(mask[Nt]> 127.0f and mask[St]< 127.0f and mask[Wt]> 127.0f and mask[Et]< 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(buf1[Nt*3+0]+buf1[Wt*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(buf1[Nt*3+1]+buf1[Wt*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(buf1[Nt*3+2]+buf1[Wt*3+2]))/4;
		}
		else if(mask[Nt]> 127.0f and mask[St]> 127.0f and mask[Wt]< 127.0f and mask[Et]< 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(buf1[Nt*3+0]+buf1[St*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(buf1[Nt*3+1]+buf1[St*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(buf1[Nt*3+2]+buf1[St*3+2]))/4;
		}
		else if(mask[Nt]< 127.0f and mask[St]< 127.0f and mask[Wt]< 127.0f and mask[Et]> 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(buf1[Et*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(buf1[Et*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(buf1[Et*3+2]))/4;
		}
		else if(mask[Nt]< 127.0f and mask[St]< 127.0f and mask[Wt]> 127.0f and mask[Et]< 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(buf1[Wt*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(buf1[Wt*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(buf1[Wt*3+2]))/4;
		}
		else if(mask[Nt]< 127.0f and mask[St]> 127.0f and mask[Wt]< 127.0f and mask[Et]< 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(buf1[St*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(buf1[St*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(buf1[St*3+2]))/4;
		}
		else if(mask[Nt]> 127.0f and mask[St]< 127.0f and mask[Wt]< 127.0f and mask[Et]< 127.0f){
			buf2[curt*3+0] = (fixed[curt*3+0]+(buf1[Nt*3+0]))/4;
			buf2[curt*3+1] = (fixed[curt*3+1]+(buf1[Nt*3+1]))/4;
			buf2[curt*3+2] = (fixed[curt*3+2]+(buf1[Nt*3+2]))/4;
		}
		else{
			buf2[curt*3+0] = (fixed[curt*3+0])/4;
			buf2[curt*3+1] = (fixed[curt*3+1])/4;
			buf2[curt*3+2] = (fixed[curt*3+2])/4;
		}
	}
}


void PoissonImageCloning(
	const float *background,
	const float *target,
	const float *mask,
	float *output,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
	) {
	// set up
	float *fixed, *buf1, *buf2;
	hipMalloc(&fixed, 3*wt*ht*sizeof(float));
	hipMalloc(&buf1, 3*wt*ht*sizeof(float));
	hipMalloc(&buf2, 3*wt*ht*sizeof(float));
	
	// initialize the iteration
	dim3 gdim(CeilDiv(wt,32), CeilDiv(ht,16)), bdim(32,16);
	CalculateFixed<<<gdim, bdim>>>(
	background, target, mask, fixed,
	wb, hb, wt, ht, oy, ox
	);
	hipMemcpy(buf1, target, sizeof(float)*3*wt*ht, hipMemcpyDeviceToDevice);
	
	// iterate
	for (int i = 0; i < 10; ++i) {
		PoissonImageCloningIteration<<<gdim, bdim>>>(fixed, mask, buf1, buf2, wt, ht);
		PoissonImageCloningIteration<<<gdim, bdim>>>(fixed, mask, buf2, buf1, wt, ht);
	}
	
	// copy the image back
	hipMemcpy(output, background, wb*hb*sizeof(float)*3, hipMemcpyDeviceToDevice);
	SimpleClone<<<gdim, bdim>>>(background, buf1, mask, output,wb, hb, wt, ht, oy, ox);
	
	// clean up
	hipFree(fixed);
	hipFree(buf1);
	hipFree(buf2);
}